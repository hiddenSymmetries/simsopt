#include "hip/hip_runtime.h"
// #include "simdhelpers.h" // import above cuda_runtime to prevent collision for rsqrt
#include <hip/hip_runtime.h>
#include <iostream>
#include "tracing.h"
#include <math.h>
#include "xtensor-python/pyarray.hpp"     // Numpy bindings
typedef xt::pyarray<double> PyArray;
#include "xtensor-python/pytensor.hpp"     // Numpy bindings
typedef xt::pytensor<double, 2, xt::layout_type::row_major> PyTensor;
using std::shared_ptr;
using std::vector;
namespace py = pybind11;

#define PARTICLES_PER_BLOCK 128

// Particle Data Structure
typedef struct particle_t {
    double state[4];
    double v_perp; // Velocity perpendicular
    double v_total;
    bool has_left;
    double dt;
    double dtmax;
    double t;
    double mu;
    double derivs[42] = {0.0};
    double x_temp[4], x_err[4];
    double s_shape[4], t_shape[4], z_shape[4];
    int i, j, k;
    double interpolation_loc[3];
    bool symmetry_exploited;
    int id;
    int step_attempt, step_accept;
} particle_t;


__host__ __device__ void shape(double x, double* shape){
    shape[0] = (1.0-x)*(2.0-x)*(3.0-x)/6.0;
    shape[1] = x*(2.0-x)*(3.0-x)/2.0;
    shape[2] = x*(x-1.0)*(3.0-x)/2.0;
    shape[3] = x*(x-1.0)*(x-2.0)/6.0;
    return;         
}

__host__  __device__ __forceinline__ void interpolate(particle_t& p, const double* __restrict__ data, double* out, const double* __restrict__ srange_arr, const double* __restrict__ trange_arr, const double* __restrict__ zrange_arr, int n){


    int ns = srange_arr[2];
    int nt = trange_arr[2];
    int nz = zrange_arr[2];

    // Need to interpolate modB, modB derivs, G, and iota

    /*
    From here it remains to perform the necessary interpolations
    As opposed to Cartesian coordinates, we don't need to monitor the surface dist via interpolation
    We also don't need to calculate the derivative of any of the interpolations
    This lets us interpolate everything in one set of nested loops 
    */

    // store interpolants in a common array, indexed the same as the columns of the quad info
    // modB, derivs of modB, G, iota

    // quad pts are indexed s t z (could be improved)
    for(int ii=0; ii<=3; ++ii){ // s grid
        if((p.i+ii) < ns){
            for(int jj=0; jj<=3; ++jj){ // theta grid           
                int wrap_j = (p.j+jj) % nt;
                for(int kk=0; kk<=3; ++kk){ // zeta grid
                    int wrap_k = (p.k+kk) % nz;
                    int row_idx = (p.i+ii)*nt*nz + wrap_j*nz + wrap_k;
                    
                    double shape_val = p.s_shape[ii]*p.t_shape[jj]*p.z_shape[kk];
                    for(int zz=0; zz<6; ++zz){
                        out[zz] += data[n*row_idx + zz]*shape_val;
                    }
                }
            }
        }

    }

}

// out contains derivatives for x , y, z, v_par, and then norm of B and surface distance interpolation
__host__  __device__ void calc_derivs(particle_t& p, double* out, double* srange_arr, double* trange_arr, double* zrange_arr, double* quadpts_arr, double m, double q, double mu, double psi0){
    /*
    * Returns     
    out[0] = ds/dtime
    out[1] = dtheta/dtime
    out[2] = dzeta/dtime

    out[3] = dvpar/dtime;
    out[4] = modB;
    */
    
    double interpolants[6] = {0.0};
    interpolate(p, quadpts_arr, interpolants, srange_arr, trange_arr, zrange_arr, 6);

    double s = sqrt(p.x_temp[0]*p.x_temp[0] + p.x_temp[1]*p.x_temp[1]);
    double theta = atan2(p.x_temp[1], p.x_temp[0]);
    double z = p.x_temp[2];
    double v_par = p.x_temp[3];
    if(p.symmetry_exploited){
        interpolants[2] *= -1.0;
        interpolants[3] *= -1.0;
    }

    double fak1 = m*v_par*v_par/interpolants[0] + m*mu;
    double sdot = -interpolants[2]*fak1 / (q*psi0);
    double tdot = interpolants[1]*fak1 / (q*psi0) + interpolants[5]*v_par*interpolants[0]/interpolants[4];

    out[0] = sdot*cos(theta) - s*sin(theta)*tdot;
    out[1] = sdot*sin(theta) + s*cos(theta)*tdot;
    out[2] = v_par*interpolants[0]/interpolants[4];
    out[3] = -(interpolants[5]*interpolants[2] + interpolants[3])*mu*interpolants[0] / interpolants[4];

    out[4] = interpolants[0]; // modB
    out[5] = interpolants[4]; // G


}



__host__ __device__ void build_state(particle_t& p, int deriv_id, double* srange_arr, double* trange_arr, double* zrange_arr){
   

    const double b1 = 35.0 / 384.0, b3 = 500.0 / 1113.0, b4 = 125.0 / 192.0, b5 = -2187.0 / 6784.0, b6 = 11.0 / 84.0;
    double wgts[6] = {0.0}; 

    for (int i = 0; i < 4; i++) {
        p.x_temp[i] = p.state[i];
    }

    switch(deriv_id){
        case 0:
            // wgts = {0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
            break;
        case 1:
            // wgts = {1.0/5.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
            wgts[0] = 1.0/5.0;
            break;
        case 2:
            // wgts = {3.0 / 40.0, 9.0 / 40.0, 0.0, 0.0, 0.0, 0.0};
            wgts[0] = 3.0 / 40.0;
            wgts[1] = 9.0 / 40.0;
            break;
        case 3:
            // wgts = {44.0 / 45.0, -56.0 / 15.0, 32.0 / 9.0, 0.0, 0.0, 0.0, 0.0};
            wgts[0] = 44.0 / 45.0;
            wgts[1] = -56.0 / 15.0;
            wgts[2] = 32.0 / 9.0;
            break;
        case 4:
            // wgts = {19372.0 / 6561.0, -25360.0 / 2187.0, 64448.0 / 6561.0, -212.0 / 729.0, 0.0, 0.0, 0.0};
            wgts[0] = 19372.0 / 6561.0;
            wgts[1] = -25360.0 / 2187.0;
            wgts[2] = 64448.0 / 6561.0;
            wgts[3] = -212.0 / 729.0;
            break;
        case 5:
            // wgts = {9017.0 / 3168.0, -355.0 / 33.0, 46732.0 / 5247.0, 49.0 / 176.0,-5103.0 / 18656.0, 0.0, 0.0};
            wgts[0] = 9017.0 / 3168.0;
            wgts[1] = -355.0 / 33.0;
            wgts[2] = 46732.0 / 5247.0;
            wgts[3] = 49.0 / 176.0;
            wgts[4] = -5103.0 / 18656.0;
            break;
        case 6:
            // wgts = {35.0 / 384.0, 0.0, 500.0 / 1113.0, 125.0 / 192.0, -2187.0 / 6784.0, 11.0 / 84.0, 0.0};
            wgts[0] = 35.0 / 384.0;
            wgts[2] = 500.0 / 1113.0;
            wgts[3] = 125.0 / 192.0; 
            wgts[4] = -2187.0 / 6784.0;
            wgts[5] = 11.0 / 84.0;
            break;
        default:
            break;
    }

    // create pt where deriv is to be computed
    for (int j=0; j<6; ++j){
        for(int i=0; i<4; ++i){
            p.x_temp[i] += p.dt * wgts[j] * p.derivs[6*j+i];
        }
    } 


    // transform to Boozer coordinates for B-field info
    double s = sqrt(p.x_temp[0]*p.x_temp[0] + p.x_temp[1]*p.x_temp[1]);
    double theta = atan2(p.x_temp[1], p.x_temp[0]);
    double z = p.x_temp[2];
    double v_par = p.x_temp[3];
    
    // we want to exploit periodicity in the B-field, but leave sine(theta) unchanged
    double t = fmod(theta, 2*M_PI);
    t += 2*M_PI*(t < 0);

    // we can modify z because it's only used to access the B-field location
    double period = zrange_arr[1];
    z = fmod(z, period);
    z += period*(z < 0);

    // exploit stellarator symmetry
    p.symmetry_exploited = t > M_PI;
    if(p.symmetry_exploited){
        z = period - z;
        t = 2*M_PI - t;
    }
    p.interpolation_loc[0] = s;
    p.interpolation_loc[1] = t;
    p.interpolation_loc[2] = z;

    /*
    * index into the grid and calculate weights
    */ 

    double s_grid_size = (srange_arr[1]-srange_arr[0]) / (srange_arr[2]-1);
    double theta_grid_size = (trange_arr[1]-trange_arr[0]) / (trange_arr[2]-1);
    double zeta_grid_size = (zrange_arr[1]-zrange_arr[0]) / (zrange_arr[2]-1);

    p.i = 3*((int) ((s - srange_arr[0]) / s_grid_size) / 3);
    p.j = 3*((int) ((t - trange_arr[0]) / theta_grid_size) / 3);
    p.k = 3*((int) ((z - zrange_arr[0]) / zeta_grid_size) / 3);

    // prevent out of bounds accesses
    p.i = min(p.i, (int)srange_arr[2]-4);
    p.j = min(p.j, (int)trange_arr[2]-4);
    p.k = min(p.k, (int)zrange_arr[2]-4);

    // normalized positions in local grid wrt e.g. r at index i
    // maps the position to [0,3] in the "meta grid"

    double s_rel = (s -  p.i*s_grid_size - srange_arr[0]) / s_grid_size;
    double theta_rel = (t -  p.j*theta_grid_size - trange_arr[0]) / theta_grid_size;
    double zeta_rel = (z - p.k*zeta_grid_size - zrange_arr[0]) / zeta_grid_size;
    
    shape(s_rel, p.s_shape);
    shape(theta_rel, p.t_shape);
    shape(zeta_rel, p.z_shape);

}


// set initial time step, calculate mu
__host__ __device__ void setup_particle(particle_t& p, double* srange_arr, double* trange_arr, double* zrange_arr, double* quadpts_arr,
                         double tmax, double m, double q, double psi0){
                             // double mu;
    p.t = 0.0;
    p.dt = 0.0;
    build_state(p, 0, srange_arr, trange_arr, zrange_arr);

    // dummy call to get norm B
    calc_derivs(p, p.derivs, srange_arr, trange_arr, zrange_arr, quadpts_arr, m, q, -1, psi0);

    double v_perp2 = p.v_perp*p.v_perp;
    double denom = 1 / (2*p.derivs[4]);
    p.mu = v_perp2 * denom;

    p.dtmax = 0.5*M_PI*abs(p.derivs[5]) / (p.derivs[4]*p.v_total);
    p.dt = 1e-3*p.dtmax;

}

__host__ __device__ void adjust_time(particle_t& p, double tmax){
    if(p.has_left){
        return;
    }

    const double bhat1 = 71.0 / 57600.0, bhat3 = -71.0 / 16695.0, bhat4 = 71.0 / 1920.0, bhat5 = -17253.0 / 339200.0, bhat6 = 22.0 / 525.0, bhat7 = -1.0 / 40.0;

    // Compute  error
    // https://live.boost.org/doc/libs/1_82_0/libs/numeric/odeint/doc/html/boost_numeric_odeint/odeint_in_detail/steppers.html
    // resolve typo in boost docs: https://numerical.recipes/book.html
    double atol=1e-9;
    double rtol=1e-9;
    double err = 0.0;
    bool accept = true;
    for (int i = 0; i < 4; i++) {
        p.x_err[i] = p.dt*(bhat1 * p.derivs[i] + bhat3 * p.derivs[12+i] + bhat4 * p.derivs[18+i] + bhat5 * p.derivs[24+i] + bhat6 * p.derivs[30+i] + bhat7 * p.derivs[36+i]);
       
        if(i==3){ // account for scale of v_par in absolute tolerance
            atol *= 1e5;
        }
        p.x_err[i] = fabs(p.x_err[i]) / (atol + rtol*(fabs(p.state[i]) + p.dt*fabs(p.derivs[i])));      
        err = fmax(err, p.x_err[i]);
    }
    // Compute new step size
    double dt_new = p.dt*0.9*pow(err, -1.0/3.0);
    dt_new = fmax(dt_new, 0.2 * p.dt);  // Limit step size reduction
    dt_new = fmin(dt_new, 5.0 * p.dt);  // Limit step size increase
    dt_new = fmin(p.dtmax, dt_new);
    if ((0.5 < err) & (err < 1.0)){
        dt_new = p.dt;
    }
    p.step_attempt++;
    if (err <= 1.0) {
        // Accept the step
        p.t += p.dt;
        p.dt = fmin(dt_new, tmax - p.t);

        p.state[0] = p.x_temp[0];
        p.state[1] = p.x_temp[1];
        p.state[2] = p.x_temp[2];
        p.state[3] = p.x_temp[3];

        double s = sqrt(p.state[0]*p.state[0] + p.state[1]*p.state[1]);
        p.has_left = s >= 1;
        p.step_accept++;


    } else {
        // Reject the step and try again with smaller dt
        p.dt = dt_new;
    }

}
__host__ __device__    void trace_particle(particle_t& p, double* srange_arr, double* trange_arr, double* zrange_arr, double* quadpts_arr,
                         double tmax, double m, double q, double psi0){

    setup_particle(p, srange_arr, trange_arr, zrange_arr, quadpts_arr, tmax, m, q, psi0);

    int counter = 0;

    while(p.t < tmax){
        for(int k=0; k<7; ++k){
            build_state(p, k, srange_arr, trange_arr, zrange_arr);
            calc_derivs(p, p.derivs + 6*k, srange_arr, trange_arr, zrange_arr, quadpts_arr, m, q, p.mu, psi0);
        }
        adjust_time(p, tmax);
        
        double s = sqrt(p.state[0]*p.state[0] + p.state[1]*p.state[1]);
        if(s >= 1){
            p.has_left = true;
            return;
        }

        counter++;

    }
    return;
}

__global__ void particle_trace_kernel(particle_t* particles, double* srange_arr, double* trange_arr, double* zrange_arr, double* quadpts_arr,
                        double tmax, double m, double q, double psi0, int nparticles){
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx < nparticles){
        trace_particle(particles[idx], srange_arr, trange_arr, zrange_arr, quadpts_arr, tmax, m, q, psi0);
    }
}


// __global__ void setup_particle_kernel(particle_t* particles, double* srange_arr, double* trange_arr, double* zrange_arr, double* quadpts_arr,
//                         double tmax, double m, double q, double psi0, int nparticles){
//     int idx = threadIdx.x + blockIdx.x*blockDim.x;
//     int particle_id = idx / 6;
//     if(particle_id < nparticles){
//         setup_particle(particles[particle_id], srange_arr, trange_arr, zrange_arr, quadpts_arr, tmax, m, q, psi0);
//     }
// }

// __global__ void build_state_kernel(particle_t* particles, int deriv_id, double* srange_arr, double* trange_arr, double* zrange_arr, int nparticles){
//     int idx = threadIdx.x + blockIdx.x*blockDim.x;
//     if(idx < nparticles){
//         build_state(particles[idx], deriv_id, srange_arr, trange_arr, zrange_arr);
//     }
// }

 
// __global__ void calc_derivs_kernel(particle_t* particles, int deriv_id, double* srange_arr, double* trange_arr, double* zrange_arr, double* quadpts_arr, double m, double q, double psi0, int nparticles){
//     int idx = threadIdx.x + blockIdx.x*blockDim.x;
//     int particle_id = idx / 6;
//     if(particle_id < nparticles){
//         calc_derivs(particles[particle_id], particles[particle_id].derivs + 6*deriv_id, srange_arr, trange_arr, zrange_arr, quadpts_arr, m, q, particles[particle_id].mu, psi0);
//     }
// }


// __global__ void count_done_kernel(particle_t* particles, double tmax, int *total_done, int nparticles){
//     int idx = threadIdx.x + blockIdx.x*blockDim.x;
//     if(idx < nparticles){
//         int is_done = (int) (particles[idx].has_left || (particles[idx].t >= tmax));
//         atomicAdd(total_done, is_done);
//     }
// }

// __global__ void adjust_time_kernel(particle_t* particles, double tmax, int nparticles){
//     int idx = threadIdx.x + blockIdx.x*blockDim.x;
//     if(idx < nparticles){
//         adjust_time(particles[idx], tmax);
//     }
// }


extern "C" vector<double> gpu_tracing(py::array_t<double> quad_pts, py::array_t<double> srange,
        py::array_t<double> trange, py::array_t<double> zrange, py::array_t<double> stz_init, double m, double q, double vtotal, py::array_t<double> vtang, 
        double tmax, double tol, double psi0, int nparticles){

    //  read data in from python
    auto ptr = stz_init.data();
    int size = stz_init.size();
    double stz_init_arr[size];
    std::memcpy(stz_init_arr, ptr, size * sizeof(double));
    
    py::buffer_info vtang_buf = vtang.request();
    double* vtang_arr = static_cast<double*>(vtang_buf.ptr);

    // contains b field
    py::buffer_info quadpts_buf = quad_pts.request();
    double* quadpts_arr = static_cast<double*>(quadpts_buf.ptr);

    py::buffer_info s_buf = srange.request();
    double* srange_arr = static_cast<double*>(s_buf.ptr);

    py::buffer_info t_buf = trange.request();
    double* trange_arr = static_cast<double*>(t_buf.ptr);

    py::buffer_info z_buf = zrange.request();
    double* zrange_arr = static_cast<double*>(z_buf.ptr);

    particle_t* particles =  new particle_t[nparticles];

    /*
    * y1 = s*cos(theta)
    * y2 = s*sin(theta)
    */

    // load initial conditions
    for(int i=0; i<nparticles; ++i){
        int start = 3*i;

        double s = stz_init_arr[start];
        double theta = stz_init_arr[start+1];
        
        // convert to alternative coordinates
        particles[i].state[0] = s*cos(theta);
        particles[i].state[1] = s*sin(theta);
        
        particles[i].state[2] = stz_init_arr[start+2];
        particles[i].state[3] = vtang_arr[i];
        particles[i].v_perp = sqrt(vtotal*vtotal -  vtang_arr[i]*vtang_arr[i]);
        particles[i].v_total = vtotal;
        particles[i].has_left = false;
        particles[i].t = 0;
        
        particles[i].step_accept = 0;
        particles[i].step_attempt = 0;
        particles[i].id = i;
        
    }
   
    
    particle_t* particles_d;
    hipMalloc((void**)&particles_d, nparticles * sizeof(particle_t));
    hipMemcpy(particles_d, particles, nparticles * sizeof(particle_t), hipMemcpyHostToDevice);

    double* srange_d;
    hipMalloc((void**)&srange_d, 3 * sizeof(double));
    hipMemcpy(srange_d, srange_arr, 3 * sizeof(double), hipMemcpyHostToDevice);

    double* zrange_d;
    hipMalloc((void**)&zrange_d, 3 * sizeof(double));
    hipMemcpy(zrange_d, zrange_arr, 3 * sizeof(double), hipMemcpyHostToDevice);

    double* trange_d;
    hipMalloc((void**)&trange_d, 3 * sizeof(double));
    hipMemcpy(trange_d, trange_arr, 3 * sizeof(double), hipMemcpyHostToDevice);


    double* quadpts_d;
    hipMalloc((void**)&quadpts_d, quad_pts.size() * sizeof(double));
    hipMemcpy(quadpts_d, quadpts_arr, quad_pts.size() * sizeof(double), hipMemcpyHostToDevice);

    int nthreads = 256;
    int nblks = nparticles / nthreads + 1;
    std::cout << "starting particle tracing kernel\n";

       
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    particle_trace_kernel<<<nblks, nthreads>>>(particles_d, srange_d, trange_d, zrange_d, quadpts_d, tmax, m, q, psi0, nparticles);

    hipMemcpy(particles, particles_d, nparticles * sizeof(particle_t), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "tracing kernels time (ms): " << milliseconds<< "\n";
    
    vector<double> particle_output(7*nparticles);
    for(int i=0; i<nparticles; ++i){
        double y1 = particles[i].state[0];
        double y2 = particles[i].state[1];
        double z = particles[i].state[2];
        double v_par = particles[i].state[3];

        // last location in Boozer coordinates
        particle_output[7*i] = sqrt(y1*y1 + y2*y2);
        particle_output[7*i + 1] = atan2(y2, y1);
        particle_output[7*i + 2] = z;
        particle_output[7*i + 3] = v_par;
        particle_output[7*i + 4] = particles[i].t;
        particle_output[7*i + 5] = particles[i].step_accept;
        particle_output[7*i + 6] = particles[i].step_attempt;
    }


    delete[] particles;

    return particle_output;
}

extern "C" py::array_t<double> test_interpolation(py::array_t<double> quad_pts, py::array_t<double> srange, py::array_t<double> trange, py::array_t<double> zrange, py::array_t<double> loc, int n){
    py::buffer_info quadpts_buf = quad_pts.request();
    double* quadpts_arr = static_cast<double*>(quadpts_buf.ptr);

    py::buffer_info s_buf = srange.request();
    double* srange_arr = static_cast<double*>(s_buf.ptr);

    py::buffer_info t_buf = trange.request();
    double* trange_arr = static_cast<double*>(t_buf.ptr);

    py::buffer_info z_buf = zrange.request();
    double* zrange_arr = static_cast<double*>(z_buf.ptr);

    py::buffer_info loc_buf = loc.request();
    double* loc_arr = static_cast<double*>(loc_buf.ptr);

    double out[n];

    // double s = loc_arr[0];
    double t = loc_arr[1];
    double z = loc_arr[2];
    // we want to exploit periodicity in the B-field, but leave sine(theta) unchanged
    t = fmod(t, 2*M_PI);
    t += 2*M_PI*(t < 0);

    // we can modify z because it's only used to access the B-field location
    double period = zrange_arr[1];
    z = fmod(z, period);
    z += period*(z < 0);

    
    // exploit stellarator symmetry
    bool symmetry_exploited = t > M_PI;
    if(symmetry_exploited){
        z = period - z;
        t = 2*M_PI - t;
    }
    loc_arr[1] = t;
    loc_arr[2] = z;

    if(symmetry_exploited){
        out[2] *= -1.0;
        out[3] *= -1.0;
    }

    auto result = py::array_t<double>(n, out);
    return result;

}

__global__ void test_gpu_interpolation_kernel(double* quad_pts, double* srange, double* trange, double* zrange, double* loc, double* out, int n, int n_points){
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx < n_points){
        double* loc_arr = loc + 3*idx;
        double* out_arr  =  out + idx*n;

        particle_t p;
        double s = loc_arr[0];
        double t = loc_arr[1];
        double z = loc_arr[2];

        p.state[0] = s*cos(t);
        p.state[1] = s*sin(t);
        p.state[2] = z;

        p.dt = 1e-3; //needed for build_state

        build_state(p, 0, srange, trange, zrange);
        
        interpolate(p, quad_pts, out_arr, srange, trange, zrange, n);

        if(p.symmetry_exploited){
            out_arr[2] *= -1.0;
            out_arr[3] *= -1.0;
        }
    }
}


extern "C" py::array_t<double> test_gpu_interpolation(py::array_t<double> quad_pts, py::array_t<double> srange, py::array_t<double> trange, py::array_t<double> zrange, py::array_t<double> loc, int n, int n_points){
    py::buffer_info quadpts_buf = quad_pts.request();
    double* quadpts_arr = static_cast<double*>(quadpts_buf.ptr);

    py::buffer_info s_buf = srange.request();
    double* srange_arr = static_cast<double*>(s_buf.ptr);

    py::buffer_info t_buf = trange.request();
    double* trange_arr = static_cast<double*>(t_buf.ptr);

    py::buffer_info z_buf = zrange.request();
    double* zrange_arr = static_cast<double*>(z_buf.ptr);

    py::buffer_info loc_buf = loc.request();
    double* loc_arr = static_cast<double*>(loc_buf.ptr);


    double* srange_d;
    hipMalloc((void**)&srange_d, 3 * sizeof(double));
    hipMemcpy(srange_d, srange_arr, 3 * sizeof(double), hipMemcpyHostToDevice);

    double* zrange_d;
    hipMalloc((void**)&zrange_d, 3 * sizeof(double));
    hipMemcpy(zrange_d, zrange_arr, 3 * sizeof(double), hipMemcpyHostToDevice);

    double* trange_d;
    hipMalloc((void**)&trange_d, 3 * sizeof(double));
    hipMemcpy(trange_d, trange_arr, 3 * sizeof(double), hipMemcpyHostToDevice);

    double* quadpts_d;
    hipMalloc((void**)&quadpts_d, quad_pts.size() * sizeof(double));
    hipMemcpy(quadpts_d, quadpts_arr, quad_pts.size() * sizeof(double), hipMemcpyHostToDevice);

    double* loc_d;
    hipMalloc((void**)&loc_d, loc.size() * sizeof(double));
    hipMemcpy(loc_d, loc_arr, loc.size() * sizeof(double), hipMemcpyHostToDevice);


    double* out_d;
    hipMalloc((void**)&out_d, n*n_points * sizeof(double));

    int nthreads = 256;
    int nblks = n_points / nthreads + 1;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    test_gpu_interpolation_kernel<<<nblks, nthreads>>>(quadpts_d, srange_d, trange_d, zrange_d, loc_d, out_d, n, n_points);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "interpolation kernel time (ms): " << milliseconds<< "\n";
    
    double out[n*n_points];
    hipMemcpy(&out, out_d, n*n_points * sizeof(double), hipMemcpyDeviceToHost);
    auto result = py::array_t<double>(n*n_points, out);
    return result;

}


__global__ void test_gpu_derivs_kernel(double* quad_pts, double* srange, double* trange, double* zrange, double* loc, double* vpar, double vtotal, double* out, double m, double q, double psi0, int n_points){
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx < n_points){
        double* loc_arr = loc + 3*idx;
        double* out_arr  =  out + 4*idx;
        double vpar_val = vpar[idx];

        particle_t p;
        double s = loc_arr[0];
        double t = loc_arr[1];
        double z = loc_arr[2];

        p.state[0] = s*cos(t);
        p.state[1] = s*sin(t);
        p.state[2] = z;
        p.state[3] = vpar_val;
        p.v_total = vtotal;
        p.v_perp = sqrt(vtotal*vtotal -  vpar_val*vpar_val);

        setup_particle(p, srange, trange, zrange, quad_pts, 1e-2, m, q, psi0);

        calc_derivs(p, p.derivs, srange, trange, zrange, quad_pts, m, q, p.mu, psi0);

        out_arr[0] = p.derivs[0];
        out_arr[1] = p.derivs[1];
        out_arr[2] = p.derivs[2];
        out_arr[3] = p.derivs[3];

    }
}

extern "C" py::array_t<double> test_derivatives(py::array_t<double> quad_pts, py::array_t<double> srange, py::array_t<double> trange, py::array_t<double> zrange, py::array_t<double> loc, py::array_t<double> vpar, double v_total, double m, double q, double psi0, int n_points){
    py::buffer_info quadpts_buf = quad_pts.request();
    double* quadpts_arr = static_cast<double*>(quadpts_buf.ptr);

    py::buffer_info s_buf = srange.request();
    double* srange_arr = static_cast<double*>(s_buf.ptr);

    py::buffer_info t_buf = trange.request();
    double* trange_arr = static_cast<double*>(t_buf.ptr);

    py::buffer_info z_buf = zrange.request();
    double* zrange_arr = static_cast<double*>(z_buf.ptr);

    py::buffer_info loc_buf = loc.request();
    double* loc_arr = static_cast<double*>(loc_buf.ptr);

    py::buffer_info vpar_buf = vpar.request();
    double* vpar_arr = static_cast<double*>(vpar_buf.ptr);
    

    double* srange_d;
    hipMalloc((void**)&srange_d, 3 * sizeof(double));
    hipMemcpy(srange_d, srange_arr, 3 * sizeof(double), hipMemcpyHostToDevice);

    double* zrange_d;
    hipMalloc((void**)&zrange_d, 3 * sizeof(double));
    hipMemcpy(zrange_d, zrange_arr, 3 * sizeof(double), hipMemcpyHostToDevice);

    double* trange_d;
    hipMalloc((void**)&trange_d, 3 * sizeof(double));
    hipMemcpy(trange_d, trange_arr, 3 * sizeof(double), hipMemcpyHostToDevice);

    double* quadpts_d;
    hipMalloc((void**)&quadpts_d, quad_pts.size() * sizeof(double));
    hipMemcpy(quadpts_d, quadpts_arr, quad_pts.size() * sizeof(double), hipMemcpyHostToDevice);

    double* loc_d;
    hipMalloc((void**)&loc_d, loc.size() * sizeof(double));
    hipMemcpy(loc_d, loc_arr, loc.size() * sizeof(double), hipMemcpyHostToDevice);

    double* vpar_d;
    hipMalloc((void**)&vpar_d, vpar.size() * sizeof(double));
    hipMemcpy(vpar_d, vpar_arr, vpar.size() * sizeof(double), hipMemcpyHostToDevice);

    double* out_d;
    hipMalloc((void**)&out_d, 4*n_points * sizeof(double));



    int nthreads = 256;
    int nblks = n_points / nthreads + 1;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    test_gpu_derivs_kernel<<<nblks, nthreads>>>(quadpts_d, srange_d, trange_d, zrange_d, loc_d, vpar_d, v_total, out_d, m, q, psi0, n_points);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "interpolation kernel time (ms): " << milliseconds<< "\n";
    
    double out[4*n_points];
    hipMemcpy(&out, out_d, 4*n_points * sizeof(double), hipMemcpyDeviceToHost);
    auto result = py::array_t<double>(4*n_points, out);
    return result;
}

__global__ void test_gpu_timestep_kernel(particle_t* particles, double* srange_arr, double* trange_arr, double* zrange_arr, double* quadpts_arr,
                        double m, double q, double psi0, int nparticles){
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    if(idx < nparticles){
        setup_particle(particles[idx], srange_arr, trange_arr, zrange_arr, quadpts_arr, 1e-2, m, q, psi0);

        while(particles[idx].t == 0.0){
            for(int k=0; k<7; ++k){
                build_state(particles[idx], k, srange_arr, trange_arr, zrange_arr);
                calc_derivs(particles[idx], particles[idx].derivs + 6*k, srange_arr, trange_arr, zrange_arr, quadpts_arr, m, q, particles[idx].mu, psi0);
            }
            adjust_time(particles[idx], 1e-2);
        }
    }
    return;
}



extern "C" vector<double> test_timestep(py::array_t<double> quad_pts, py::array_t<double> srange,
        py::array_t<double> trange, py::array_t<double> zrange, py::array_t<double> stz_init, double m, double q, double vtotal, py::array_t<double> vtang, 
        double tol, double psi0, int nparticles){

    //  read data in from python
    auto ptr = stz_init.data();
    int size = stz_init.size();
    double stz_init_arr[size];
    std::memcpy(stz_init_arr, ptr, size * sizeof(double));

    py::buffer_info vtang_buf = vtang.request();
    double* vtang_arr = static_cast<double*>(vtang_buf.ptr);

    // contains b field
    py::buffer_info quadpts_buf = quad_pts.request();
    double* quadpts_arr = static_cast<double*>(quadpts_buf.ptr);

    py::buffer_info s_buf = srange.request();
    double* srange_arr = static_cast<double*>(s_buf.ptr);

    py::buffer_info t_buf = trange.request();
    double* trange_arr = static_cast<double*>(t_buf.ptr);

    py::buffer_info z_buf = zrange.request();
    double* zrange_arr = static_cast<double*>(z_buf.ptr);


    particle_t* particles =  new particle_t[nparticles];

    // convert to alternative coordinates
    /*
    * y1 = s*cos(theta)
    * y2 = s*sin(theta)
    */

    // load initial conditions
    for(int i=0; i<nparticles; ++i){
        int start = 3*i;

        double s = stz_init_arr[start];
        double theta = stz_init_arr[start+1];
        
        // convert to alternative coordinates
        particles[i].state[0] = s*cos(theta);
        particles[i].state[1] = s*sin(theta);
        
        particles[i].state[2] = stz_init_arr[start+2];
        particles[i].state[3] = vtang_arr[i];
        particles[i].v_perp = sqrt(vtotal*vtotal -  vtang_arr[i]*vtang_arr[i]);
        particles[i].v_total = vtotal;
        particles[i].has_left = false;
        particles[i].t = 0;
        
        particles[i].step_accept = 0;
        particles[i].step_attempt = 0;
        particles[i].id = i;
    }
    
    particle_t* particles_d;
    hipMalloc((void**)&particles_d, nparticles * sizeof(particle_t));
    hipMemcpy(particles_d, particles, nparticles * sizeof(particle_t), hipMemcpyHostToDevice);

    double* srange_d;
    hipMalloc((void**)&srange_d, 3 * sizeof(double));
    hipMemcpy(srange_d, srange_arr, 3 * sizeof(double), hipMemcpyHostToDevice);

    double* zrange_d;
    hipMalloc((void**)&zrange_d, 3 * sizeof(double));
    hipMemcpy(zrange_d, zrange_arr, 3 * sizeof(double), hipMemcpyHostToDevice);

    double* trange_d;
    hipMalloc((void**)&trange_d, 3 * sizeof(double));
    hipMemcpy(trange_d, trange_arr, 3 * sizeof(double), hipMemcpyHostToDevice);


    double* quadpts_d;
    hipMalloc((void**)&quadpts_d, quad_pts.size() * sizeof(double));
    hipMemcpy(quadpts_d, quadpts_arr, quad_pts.size() * sizeof(double), hipMemcpyHostToDevice);

    int nthreads = 256;
    int nblks = nparticles / nthreads + 1;
    std::cout << "starting particle tracing kernel\n";

       
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    test_gpu_timestep_kernel<<<nblks, nthreads>>>(particles_d, srange_d, trange_d, zrange_d, quadpts_d, m, q, psi0, nparticles);

    hipMemcpy(particles, particles_d, nparticles * sizeof(particle_t), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "tracing kernels time (ms): " << milliseconds<< "\n";

    
    vector<double> particle_output(7*nparticles);
    for(int i=0; i<nparticles; ++i){
        double y1 = particles[i].state[0];
        double y2 = particles[i].state[1];
        double z = particles[i].state[2];
        double v_par = particles[i].state[3];

        double s = sqrt(y1*y1 + y2*y2);
        double theta = atan2(y2, y1);
        
        // last location in Boozer coordinates
        particle_output[7*i] = s;
        particle_output[7*i + 1] = theta;
        particle_output[7*i + 2] = z;
        particle_output[7*i + 3] = v_par;
        particle_output[7*i + 4] = particles[i].t;
        particle_output[7*i + 5] = particles[i].step_accept;
        particle_output[7*i + 6] = particles[i].step_attempt;
    }


    delete[] particles;

    return particle_output;
}